
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__global__ void gpu_add(int* in, int* out, int size) {
    int id = threadIdx.x;
    out[id] = in[id] + in[size-1 - id];
    return;
}

__global__ void gpu_extra(int in, int* out) {
    clock_t start_clock = clock();
    clock_t curr;
    float seconds = 2000;
 
    while (((clock() - start_clock) / (float)CLOCKS_PER_SEC) < seconds) { 
        curr = clock(); 
    }
    return;
}

__global__ void gpu_mult(int* in, int size) {
    if (size < 1) return;
    int base = 1;
    for (int i = 0; i < size; i++) {
        base = base * in[i];
    }
    printf("MULT: %d\n", base);
    return;
}
