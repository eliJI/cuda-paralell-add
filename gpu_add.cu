
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__global__ void gpu_add(int* in, int* out, int size) {
    int id = threadIdx.x;
    out[id] = in[id] + in[size-1 - id];
}

__global__ void gpu_extra(int in, int* out) {
    clock_t start_clock = clock();
    clock_t curr;
    float seconds = 2000;
    
    while (((clock() - start_clock) / (float)CLOCKS_PER_SEC) < seconds) { 
        curr = clock(); 
    }
}
