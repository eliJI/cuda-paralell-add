
#include <hip/hip_runtime.h>
__global__ void gpu_add(int* in, int* out, int size) {
    int id = threadIdx.x;
    out[id] = in[id] + in[size-1 - id];
}
