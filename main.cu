#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void gpu_add(int*, int*, int);
__global__  void gpu_extra(int in, int* out);

int main(int argc, char** argv) {
    printf("beginning initialization\n");
    int* array;
    int* d_array;
    hipStream_t stream1, stream2;
    int* out = (int*)malloc(sizeof(int)*16);
    hipHostMalloc(&array, sizeof(int)*16, hipHostMallocDefault);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    //initialize arra y values
    for (int i = 0; i < 16; i++) {
        array[i] = 1;
    }


    int size = 16;
    int extra_out;

    printf("initializing\n");
    while (size > 1) {

        hipMalloc((void**)&d_array, sizeof(int)*size);
        hipMemcpyAsync(d_array, array, sizeof(int)*size, hipMemcpyHostToDevice, stream1);
        int* d_out;
        size = size / 2;
        hipMalloc((void**)&d_out, sizeof(int)*size);
        gpu_add<<<1,size, 0, stream1>>>(d_array, d_out, size*2);
        gpu_extra<<<1,1,0,stream2>>>(size, &extra_out);
        hipStreamSynchronize(stream1);
        hipMemcpyAsync(array, d_out,sizeof(int)*size,hipMemcpyDeviceToHost, stream1);
        
        //print intermediate
        printf("intermediate sum:\n");
        for(int i = 0; i < size; i++) {
            printf("%d, ", array[i]);
        }
        printf("\n");
    }

    //flag to keep track of kernel launches
    hipStreamSynchronize(stream2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    return 0;
 
}
