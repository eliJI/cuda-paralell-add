#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void gpu_add(int*, int*, int);
__global__  void gpu_extra(int, int*);
__global__ void gpu_mult(int*, int);

int main(int argc, char** argv) {
    printf("beginning initialization\n");
    int* array;
    int* d_array;
    int* d2_array;

    hipStream_t stream1, stream2;
    int* out = (int*)malloc(sizeof(int)*16);
    hipHostMalloc(&array, sizeof(int)*16, hipHostMallocDefault);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    //initialize arra y values
    for (int i = 0; i < 16; i++) {
        array[i] = 1;
    }


    int size = 16;
    int extra_out;

    printf("initializing\n");
    while (size > 1) {

        hipMalloc((void**)&d_array, sizeof(int)*size);
        hipMemcpyAsync(d_array, array, sizeof(int)*size, hipMemcpyHostToDevice, stream1);
        int* d_out;
        size = size / 2;
        hipMalloc((void**)&d_out, sizeof(int)*size);
        hipMalloc((void**)&d2_array, sizeof(int)*size);
        gpu_add<<<1, size, 0, stream1>>>(d_array, d_out, size*2);
        //gpu_extra<<<1,1,0,stream2>>>(size, &extra_out);
        hipMemcpyAsync(array, d_out,sizeof(int)*size,hipMemcpyDeviceToHost, stream1);
        hipStreamSynchronize(stream1);

        hipMemcpyAsync(d2_array, array, sizeof(int)*size, hipMemcpyHostToDevice, stream2);
        gpu_mult<<<1, 1, 0, stream2>>>(d2_array, size);
        
        //print intermediate
        printf("intermediate sum:\n");
        for(int i = 0; i < size; i++) {
            printf("%d, ", array[i]);
        }
        printf("\n");

        hipStreamSynchronize(stream2);
    }

    //flag to keep track of kernel launches
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    return 0;
 
}
