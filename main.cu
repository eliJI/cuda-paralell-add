#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void gpu_add(int*, int*, int);

int main(int argc, char** argv) {
    printf("beginning initialization\n");
    int* array;
    int* d_array;
    int* out = (int*)malloc(sizeof(int)*16);
    array = (int*)malloc(sizeof(int)*16);

    //initialize arra y values
    for (int i = 0; i < 16; i++) {
        array[i] = 1;
    }


    int size = 16;

    printf("initializing\n");
    while (size > 1) {

        hipMalloc((void**)&d_array, sizeof(int)*size);
        hipMemcpyAsync(d_array, array, sizeof(int)*size, hipMemcpyHostToDevice);
        int* d_out;
        size = size / 2;
        hipMalloc((void**)&d_out, sizeof(int)*size);
        gpu_add<<<1,size>>>(d_array, d_out, size*2);
        hipDeviceSynchronize();
        hipMemcpyAsync(array, d_out,sizeof(int)*size,hipMemcpyDeviceToHost);
        
        //print intermediate
        printf("intermediate sum:\n");
        for(int i = 0; i < size; i++) {
            printf("%d, ", array[i]);
        }
        printf("\n");
    }

    //flag to keep track of kernel launches

    return 0;
 
}
